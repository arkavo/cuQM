#include "cuQMstd.cuh"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <string>
//#include "hip/hip_runtime.h"

int main(int argc, char* argv[])
{
    SPACE sample1;
    int threads = 4;
    if(argc==2)
    {
        threads = std::stoi(argv[1]);
    }
    sample1.initialize(10,1,1);
    
    double* space_arr;
    double* sampleARR;
    space_arr = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    sampleARR = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                *(space_arr + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y) = 1.0; //double(i+j+k -i*j*k);
                *(sampleARR + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y)= 2.0;
                //std::cout<<space_arr[i][j][k]<<'\n';
            }
        }
    }
    /*
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                std::cout<<*(space_arr + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y)<<"\n";
            }
        }
    }
    std::cout<<'\n';
    */
    /*std::cout<<sample1.ADDRESS<<'\n'
    <<sample1.V<<'\n'
    <<sample1.Y<<'\n'
    <<sample1.dY<<'\n'
    <<sample1.ddY<<'\n';   
    */
    auto hst_st = std::chrono::high_resolution_clock::now();
    
    sample1.assign(sampleARR,space_arr);
    
    //hipMemcpy(sample1.ADDRESS,space_arr,sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double),hipMemcpyHostToDevice);
    
    sample1.display();
    
    //hipMemcpy(sampleARR,sample1.ADDRESS,sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double),hipMemcpyDeviceToHost);
    sample1.calx(threads,5.,space_arr);
    
    auto hst_en = std::chrono::high_resolution_clock::now();
    
    sample1.display();
    
    std::chrono::duration<float> duration = hst_en-hst_st;
    std::cout<<"\nDuration: "<<duration.count()<<"\n";

    std::cout<<"Sizes X Y Z "<<sample1.SIZE_X<<sample1.SIZE_Y<<sample1.SIZE_Z<<'\n';
    /*
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                std::cout<<*(sampleARR + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y);
            }
        }
    }
    */
    //insanely fast, print is the bootle neck
    std::cout<<*(sampleARR+sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z -1)<<'\n';
    std::cout<<'\n';
    return 0;
}