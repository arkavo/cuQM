#include "cuQMstd.cuh"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <string>
//#include "hip/hip_runtime.h"

int main(int argc, char* argv[])
{
    SPACE sample1;
    int threads = 4;
    if(argc==2)
    {
        threads = std::stoi(argv[1]);
    }
    sample1.initialize(10000,1,1);
    
    double* space_arr;
    double* sampleARR;
    
    space_arr = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    sampleARR = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                *(space_arr + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y) = 1.0; //double(i+j+k -i*j*k);
                *(sampleARR + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y)= 2.0;
            }
        }
    }
    
    auto hst_st = std::chrono::high_resolution_clock::now();
    
    sample1.assign(sampleARR,space_arr);
    
    sample1.display();

    sample1.calx(threads,5.);
    
    auto hst_en = std::chrono::high_resolution_clock::now();
    
    sample1.display();
    
    std::chrono::duration<float> duration = hst_en-hst_st;
    std::cout<<"\nDuration: "<<duration.count()<<"\n";

    std::cout<<"Sizes X Y Z "<<sample1.SIZE_X<<sample1.SIZE_Y<<sample1.SIZE_Z<<'\n';
    
    //insanely fast, print is the bootle neck
    std::cout<<*(sampleARR+sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z -1)<<'\n';
    std::cout<<'\n';
    return 0;
}