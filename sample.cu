#include "cuQMstd.cuh"
#include <iostream>
//#include "hip/hip_runtime.h"

int main()
{
    SPACE sample1;
    
    sample1.initialize(100,1,1);
    
    double* space_arr;
    double* sampleARR;
    space_arr = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    sampleARR = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                *(space_arr + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y) = double(i+j+k -i*j*k);
                //std::cout<<space_arr[i][j][k]<<'\n';
            }
        }
    }
    /*
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                std::cout<<*(space_arr + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y)<<"\n";
            }
        }
    }
    std::cout<<'\n';
    */
    /*std::cout<<sample1.ADDRESS<<'\n'
    <<sample1.V<<'\n'
    <<sample1.Y<<'\n'
    <<sample1.dY<<'\n'
    <<sample1.ddY<<'\n';   
    */
    //sample1.assign(space_arr);
    hipMemcpy(sample1.ADDRESS,space_arr,sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double),hipMemcpyHostToDevice);
    
    //sample1.display();
    hipMemcpy(sampleARR,sample1.ADDRESS,sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double),hipMemcpyDeviceToHost);
    
    std::cout<<"Sizes X Y Z "<<sample1.SIZE_X<<sample1.SIZE_Y<<sample1.SIZE_Z<<'\n';
    /*
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                std::cout<<*(sampleARR + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y);
            }
        }
    }
    */
    //insanely fast, print is the bootle neck
    std::cout<<*(sampleARR+1000000-1)<<'\n';
    std::cout<<'\n';
    return 0;
}