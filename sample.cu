#include "cuQMstd.cuh"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <string>
#include <fstream>
//#include "hip/hip_runtime.h"

int main(int argc, char* argv[])
{
    std::ofstream file;
    file.open("data.csv");
    SPACE sample1;
    int threads = 4;
    int x = 1000;
    int y = 1;
    int z = 1;
    
    if(argc==5)
    {
        x = std::stoi(argv[1]);
        y = std::stoi(argv[2]);
        z = std::stoi(argv[3]);
        threads = std::stoi(argv[4]);
    }
    
    sample1.initialize(x,y,z);
    
    double* space_arr;
    double* sampleARR;
    
    space_arr = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    sampleARR = (double*)malloc(sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double));
    
    for(int i=0;i<sample1.SIZE_X;i++)
    {
        for(int j=0;j<sample1.SIZE_Y;j++)
        {
            for(int k=0;k<sample1.SIZE_Z;k++)
            {
                *(space_arr + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y) = 200.0; //double(i+j+k -i*j*k);
                *(sampleARR + i + j*sample1.SIZE_X + k*sample1.SIZE_X*sample1.SIZE_Y)= 100.0;
            }
        }
    }
    
    auto hst_st = std::chrono::high_resolution_clock::now();
    
    sample1.assign(sampleARR,space_arr);
    
    //sample1.display();

    sample1.calx(threads,10.5);
    
    auto hst_en = std::chrono::high_resolution_clock::now();
    
    sample1.display();
    
    hipMemcpy(sampleARR,sample1.Y,sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z*sizeof(double),hipMemcpyDeviceToHost);
    
    std::chrono::duration<float> duration = hst_en-hst_st;
    std::cout<<"\nDuration: "<<duration.count()<<"\n";

    std::cout<<"Sizes X Y Z "<<sample1.SIZE_X<<sample1.SIZE_Y<<sample1.SIZE_Z<<'\n';
    
    //insanely fast, print is the bootle neck
    
    std::cout<<*(sampleARR+sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z -1)<<'\n';
    std::cout<<'\n';
    
    for(int i=0;i<sample1.SIZE_X*sample1.SIZE_Y*sample1.SIZE_Z;i++)
    {
        file<<*(sampleARR+i)<<'\n';
    }
    sample1.memclear();

    return 0;
}