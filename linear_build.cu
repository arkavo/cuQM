#include "hip/hip_runtime.h"
#include "cuQMstd.cuh"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <string>
#include <fstream>


void build(double* Y,double* V,double step,double tol, int size)
{
    bool ct = false;
    double E = 0.02;
    while(ct==false)
    {
        *Y = 100;
        *(Y+1) = (E - *V)**Y*step*step;
        for(int i=1;i<size-1;i++)
        {
            *(Y+i+1) = (E - *(V+i))* *(Y+i)*step*step + *(Y+i-1);
        }
        std::cout<<"Final value: "<<*(Y+size-1)<<" Eigen Energy: "<<E<<'\n';
        if(abs(*Y - *(Y+size-1))< tol)
        {
            ct = true;
            std::cout<<E<<'\n';
        }
        E += *(Y+size-1)*step;
    }
}

int main()
{
    std::ofstream file;
    file.open("data.csv");
    int L = 500;
    
    double* space_arr;
    double* sampleARR;
    
    space_arr = (double*)malloc(L*sizeof(double));
    sampleARR = (double*)malloc(L*sizeof(double));
    
    for(int i=0;i<L;i++)
    {
        *(space_arr+i) = 0.;
    }

    build(sampleARR,space_arr,0.05,0.005,L);
    
    for(int i=0;i<L;i++)
    {
        file<<*(sampleARR+i)<<'\n';
    }
}