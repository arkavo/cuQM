
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <string>
#include <fstream>
#include <cmath>

double* wavefxn(int L, int n)
{
    double pi = 3.14159;
    double* V = (double*)malloc(L*sizeof(double));
    for(int i=0;i<L;i++)
    {
        *(V+i) = sin(2*pi*n*i/L);
    }
    return V;
}

void display(double* parameter,int L)
{
    for(int i=0;i<L;i++)
    {
        std::cout<<*(parameter+i)<<" ";
    }
    std::cout<<" ";
}

int main()
{
    int L = 100;
    int n = 1;
    double* res;
    res = wavefxn(L,n);
    display(res,L);
    return 0;
}