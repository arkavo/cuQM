
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>
#include <chrono>
#include <string>
#include <limits.h>
#include <fstream>

using namespace std;

struct vector3
{
    float x;
    float y;
    float z;
};

class particle
{
    public:
        float mass;
        float charge;
        vector3 position;
        vector3 velocity;
};

void set_energy(double* dest,int L,double E)
{
    for(int i=0;i<L;i++)
    {
        *(dest+i) = E;
    }
}
void display(double* dest,int L)
{
    for(int i=0;i<L;i++)
    {
        std::cout<<*(dest+i)<<"\n";
    }
}

__global__ void SETUP(double* dest, double val)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *(dest+idx) = val;
}

__global__ void DERIVATIVE_STEP(double* y, double* ddy, double* V, double* E,double step, int L)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if((idx>0) && (idx<(L-1)))
    {
        *(ddy+idx) = (*(y+1+idx)-*(y-1+idx))/(2* (step)) + *(V+idx) - (*E);
    }
    else
    {
        *(ddy+idx) = (*(V+idx) - *E) * (step);
    }
}



int main(int argc,char* argv[])
{
    int L = stoi(argv[1]);    
    L = 10;
    int blocks = 2;
    int threads = 5;
    std::cout<<"Length: "<<L<<"\n";
    particle test;
    test.mass = 0;
    cout<<test.mass<<endl;

    unsigned long long SIZE_0 = ((int)sizeof(double)*L);
    double* V_HST;
    double* V_DEV;
    double Ev = 3.0;
    double* E;
    double step = 0.2;
    double* Y_DEV;
    double* dY_DEV;
    double* ddY_DEV;
    double* Y_Final;
    //setup
    int loops = 0;
    V_HST = (double*)malloc(SIZE_0);
    Y_Final = (double*)malloc(SIZE_0);
    
    hipMalloc((void**)&V_DEV,SIZE_0);
    hipMalloc((void**)&Y_DEV,SIZE_0);
    hipMalloc((void**)&dY_DEV,SIZE_0);
    hipMalloc((void**)&ddY_DEV,SIZE_0);
    hipMalloc((void**)&E,sizeof(double));
    
    set_energy(V_HST,L,2.3);
    hipMemcpy(V_DEV,V_HST,SIZE_0,hipMemcpyHostToDevice);
    hipMemcpy(E,&Ev,int(sizeof(double)),hipMemcpyHostToDevice);
    
    SETUP <<<blocks,threads>>> (Y_Final,0.);
    SETUP <<<blocks,threads>>> (V_DEV,0.); 
    SETUP <<<blocks,threads>>> (Y_DEV,0.);
    SETUP <<<blocks,threads>>> (dY_DEV,0.);
    SETUP <<<blocks,threads>>> (ddY_DEV,0.);
    
    while(loops>0)
    {
        loops-=1;
        std::cout<<loops<<"\n";
        DERIVATIVE_STEP <<<blocks,threads>>> (ddY_DEV,Y_DEV,V_DEV,E,step,L);
    }
    hipMemcpy(Y_Final,ddY_DEV,SIZE_0,hipMemcpyDeviceToHost);
    std::cout<<"\n";
    display(Y_Final,L);
    return 0;
}