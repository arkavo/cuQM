
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>
#include <chrono>
#include <string>
#include <limits.h>
#include <fstream>

using namespace std;

struct vector3
{
    float x;
    float y;
    float z;
};

class particle
{
    public:
        float mass;
        float charge;
        vector3 position;
        vector3 velocity;
};

void set_energy(double* dest,int L,double E)
{
    for(int i=0;i<L;i++)
    {
        *(dest+i) = E;
    }
}
void display(double* dest,int L)
{
    for(int i=0;i<L;i++)
    {
        std::cout<<*(dest+i)<<"\n";
    }
}

__global__ void ASSIGN(double* dest, double val)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *(dest + idx) = val;
}

__global__ void DERIVATIVE_STEP(double* y, double* ddy, double* V, double* E,double step, int L)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if((idx>0) && (idx<(L-1)))
    {
        *(ddy+idx) = (*(y+1+idx)-*(y-1+idx))/(2* (-1/step)) + *(V+idx) - (*E);
    }
    else
    {
        *(ddy+idx) = (*(V+idx) - *E) * (step);
    }
}
__global__ void UPDATE_STEP(double* Y,double* dY, double* ddY,double step,int L)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if((idx>0)&&(idx<(L-1)))
    {
        *(dY+idx) += (*(ddY+idx+1)+*(ddY+idx-1))*step/2.;
    }
    else
    {
        *(dY+idx) += *(ddY+idx) * step;
    }
}
__global__ void FINAL_STEP(double* Y,double* dY,double step,int L)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *(Y+idx) = *(dY+idx) * step;
}

__global__ void NORMALIZE_CONSTANT(double* Y,double* res)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *res += pow(*(Y+idx),2);
}

__global__ void NORMALIZE_FUNCTION(double* Y, double* res)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *(Y+idx) = *(Y+idx) / *res; 
}

int main(int argc,char* argv[])
{
    std::ofstream file;
    file.open("data.csv");
    
    int L = stoi(argv[1]);    
    int threads = stoi(argv[2]);
    int blocks = int(L/threads);
    //int threads = 5;
    std::cout<<"Length: "<<L<<"\n";
    
    unsigned long long SIZE_0 = ((int)sizeof(double)*L);
    double* V_HST;
    double* V_DEV;
    double Ev = -1.5;
    double* E;
    double step = -0.05;
    double* Y_DEV;
    double* dY_DEV;
    double* ddY_DEV;
    double* Y_Final;
    //setup
    int loops = 80000;
    V_HST = (double*)malloc(SIZE_0);
    Y_Final = (double*)malloc(SIZE_0);
    
    hipMalloc((void**)&V_DEV,SIZE_0);
    hipMalloc((void**)&Y_DEV,SIZE_0);
    hipMalloc((void**)&dY_DEV,SIZE_0);
    hipMalloc((void**)&ddY_DEV,SIZE_0);
    hipMalloc((void**)&E,sizeof(double));
    
    set_energy(V_HST,L,-3.0);
    *V_HST = 0;
    *(V_HST+L-1) = 0;
    hipMemcpy(V_DEV,V_HST,SIZE_0,hipMemcpyHostToDevice);
    hipMemcpy(E,&Ev,int(sizeof(double)),hipMemcpyHostToDevice);
    
    ASSIGN <<<blocks,threads>>> (V_DEV, 0.); 
    ASSIGN <<<blocks,threads>>> (Y_DEV, 0.);
    ASSIGN <<<blocks,threads>>> (dY_DEV, 0.);
    ASSIGN <<<blocks,threads>>> (ddY_DEV, 0.);
    
    while(loops>0)
    {
        loops-=1;
        //std::cout<<loops<<"\n";
        DERIVATIVE_STEP <<<blocks,threads>>> (Y_DEV,ddY_DEV,V_DEV,E,step,L);
        UPDATE_STEP <<<blocks,threads>>> (Y_DEV,dY_DEV,ddY_DEV,step,L);
        FINAL_STEP <<<blocks,threads>>> (Y_DEV,dY_DEV,step,L);
    }
    hipMemcpy(Y_Final,Y_DEV,SIZE_0,hipMemcpyDeviceToHost);
    std::cout<<"\n";
    display(Y_Final,L);
    for(int i=0;i<L;i++)
    {
        file<<Y_Final[i]<<"\n";
    }
    hipFree(V_DEV);
    hipFree(Y_DEV);
    hipFree(dY_DEV);
    hipFree(ddY_DEV);
    
    return 0;
}