#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>
#include <chrono>
#include <string>
#include <limits.h>
#include <fstream>

using namespace std;

struct vector3
{
    float x;
    float y;
    float z;
};

class particle
{
    public:
        float mass;
        float charge;
        vector3 position;
        vector3 velocity;
};

void set_energy(double* dest,int L,double E)
{
    for(int i=0;i<L;i++)
    {
        *(dest+i) = E;
    }
}
void display(double* dest,int L)
{
    for(int i=0;i<L;i++)
    {
        std::cout<<*(dest+i)<<"\n";
    }
}

__global__ void SETUP(double* dest, double val)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *(dest+idx) = val;
}

__global__ void DERIVATIVE_1(double* y, double* ddy, double* V, double* E,double step, int L)
{
    idx = threadIdx.x + blockIdx.x*blockDim.x;
    {
        if idx>0 && idx<(L-1)
        *(dy+idx) = (*(y+1+idx)-*(y-1+idx))/(2*step);
    }
}


int main(int argc,char* argv[])
{
    int L = stoi(argv[1]);    
    std::cout<<"Length: "<<L<<"\n";
    particle test;
    test.mass = 0;
    cout<<test.mass<<endl;

    unsigned long long SIZE_0 = ((int)sizeof(double)*L);
    double* V_HST;
    double* V_DEV;
    
    
    double* Y_DEV;
    double* dY_DEV;
    double* ddY_DEV;
    double* Y_Final;
    //setup
    int loops = 100;
    V_HST = (double*)malloc(SIZE_0);
    Y_Final = (double*)malloc(SIZE_0);
    hipMalloc((void**)&V_DEV,SIZE_0);
    hipMalloc((void**)&Y_DEV,SIZE_0);
    hipMalloc((void**)&dY_DEV,SIZE_0);
    hipMalloc((void**)&ddY_DEV,SIZE_0);
    
    set_energy(V_HST,L,2.2);
    hipMemcpy(V_DEV,V_HST,SIZE_0,hipMemcpyHostToDevice);
    SETUP <<<4,4>>> (Y_DEV,0);
    SETUP <<<4,4>>> (Y_DEV,0);
    SETUP <<<4,4>>> (Y_DEV,0);
    SETUP <<<4,4>>> (Y_DEV,0);
    
    while(loops>0)
    {

    }

    display(V_HST,L);
    return 0;
}