
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <time.h>
#include <chrono>
#include <string>
#include <limits.h>
#include <fstream>

using namespace std;

struct vector3
{
    float x;
    float y;
    float z;
};

class particle
{
    public:
        float mass;
        float charge;
        vector3 position;
        vector3 velocity;
};

void set_energy(double* dest,int L,double E)
{
    for(int i=0;i<L;i++)
    {
        *(dest+i) = E;
    }
}
void display(double* dest,int L)
{
    for(int i=0;i<L;i++)
    {
        std::cout<<*(dest+i)<<"\n";
    }
}

__global__ void ASSIGN(double* dest, double val)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *(dest + idx) = val;
}

__global__ void DERIVATIVE_STEP(double* y, double* ddy, double* V, double* E,double step, int L)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if((idx>0) && (idx<(L-1)))
    {
        *(ddy+idx) = (*(y+1+idx)-*(y-1+idx))/(2* (step)) + *(V+idx) - (*E);
    }
    else
    {
        *(ddy+idx) = (*(V+idx) - *E) * (step);
    }
}
__global__ void UPDATE_STEP(double* Y,double* dY, double* ddY,double step,int L)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if((idx>0)&&(idx<(L-1)))
    {
        *(dY+idx) += (*(ddY+idx+1)+*(ddY+idx-1))*step/2.;
    }
    else
    {
        *(dY+idx) += *(ddY+idx) * step;
    }
}
__global__ void FINAL_STEP(double* Y,double* dY,double step,int L)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    *(Y+idx) = *(dY+idx) * step;
}


int main(int argc,char* argv[])
{
    int L = stoi(argv[1]);    
    int threads = stoi(argv[2]);
    int blocks = int(L/threads);
    //int threads = 5;
    std::cout<<"Length: "<<L<<"\n";
    
    unsigned long long SIZE_0 = ((int)sizeof(double)*L);
    double* V_HST;
    double* V_DEV;
    double Ev = 3.0;
    double* E;
    double step = 0.2;
    double* Y_DEV;
    double* dY_DEV;
    double* ddY_DEV;
    double* Y_Final;
    //setup
    int loops = 1000;
    V_HST = (double*)malloc(SIZE_0);
    Y_Final = (double*)malloc(SIZE_0);
    
    hipMalloc((void**)&V_DEV,SIZE_0);
    hipMalloc((void**)&Y_DEV,SIZE_0);
    hipMalloc((void**)&dY_DEV,SIZE_0);
    hipMalloc((void**)&ddY_DEV,SIZE_0);
    hipMalloc((void**)&E,sizeof(double));
    
    set_energy(V_HST,L,2.3);
    hipMemcpy(V_DEV,V_HST,SIZE_0,hipMemcpyHostToDevice);
    hipMemcpy(E,&Ev,int(sizeof(double)),hipMemcpyHostToDevice);
    
    //ASSIGN <<<blocks,threads>>> (Y_Final, 0.1);
    ASSIGN <<<blocks,threads>>> (V_DEV, 0.); 
    ASSIGN <<<blocks,threads>>> (Y_DEV, 0.);
    ASSIGN <<<blocks,threads>>> (dY_DEV, 0.);
    ASSIGN <<<blocks,threads>>> (ddY_DEV, 0.2);
    
    while(loops>0)
    {
        loops-=1;
        //std::cout<<loops<<"\n";
        DERIVATIVE_STEP <<<blocks,threads>>> (Y_DEV,ddY_DEV,V_DEV,E,step,L);
        UPDATE_STEP <<<blocks,threads>>> (Y_DEV,dY_DEV,ddY_DEV,step,L);
        FINAL_STEP <<<blocks,threads>>> (Y_DEV,dY_DEV,step,L);
    }
    hipMemcpy(Y_Final,Y_DEV,SIZE_0,hipMemcpyDeviceToHost);
    std::cout<<"\n";
    display(Y_Final,L);
    return 0;
}